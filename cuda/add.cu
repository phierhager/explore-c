#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(int* A, int* B, int* C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread index
    if (idx < N) {  // Ensure we do not access out of bounds
        C[idx] = A[idx] + B[idx];  // Perform element-wise addition
    }
}

int main() {
    const int N = 1 << 20;  // Size of vectors
    size_t size = N * sizeof(int);

    // Allocate memory on the host
    int *h_A = new int[N];
    int *h_B = new int[N];
    int *h_C = new int[N];

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate memory on the device
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 dimGrid((N + 255) / 256);  // Number of blocks in the grid (rounded up to cover N elements)
    dim3 dimBlock(256);              // Number of threads per block (256 threads per block)
    vectorAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cout << "Error at index " << i << ": " << h_C[i] << std::endl;
            return -1;
        }
    }

    std::cout << "Vector addition completed successfully!" << std::endl;

    // Free memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
